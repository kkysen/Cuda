#include "hip/hip_runtime.h"
#include "cuda.cu"

template <typename T, uint blockSize>
__global__ void scanSingleBlock(T *a, const uint n, T *b) {
    __shared__ T s[blockSize];
    const uint i = threadIdx.x;
    if (i == blockSize) {
        b[0] = 0;
        return;
    }

    s[i] = i < n ? a[i] : 0;
    __syncthreads();

    #pragma unroll
    for (uint d = 1; d < blockSize; d <<= 1) {
        const uint j = i + d;
        if (j < blockSize) {
            s[j] += s[i];
            __syncthreads();
        }
    }
    __syncthreads();

    b[i + 1] = s[i];
}

template <typename T, uint blockSize>
__global__ void scanBlocks(T *in, T *out, T *blockSums, const uint numBlocks) {
    __shared__ T s[blockSize];
    const uint blockId = blockId();
    if (blockId >= numBlocks) {
        printf("blockId too high\n");
        return;
    }
    const uint i = threadIdx.x;
    const uint index = i + (blockId * blockSize);

    s[i] = in[index];
    __syncthreads();

    #pragma unroll
    for (uint d = 1; d < blockSize; d <<= 1) {
        const uint j = i + d;
        if (j < blockSize) {
            s[j] += s[i];
        }
        __syncthreads();
    }

    if (i == blockSize - 1) {
        out[index - i] = 0;
        blockSums[blockId] = s[i];
    } else {
        out[index + 1] = s[i];
    }
}

template <typename T, uint blockSize>
void scanBlocks(T *in, T *out, T *blockSums, const uint numBlocks, const dim3 grid) {
    std::cout << "scanning " << numBlocks << " blocks" << std::endl;
    scanBlocks<T, blockSize><<<grid, blockSize>>>(in, out, blockSums, numBlocks);
    check();
    sync();
}

// TODO make blockSums into constant or texture memory
template <typename T, uint blockSize>
__global__ void addBlockSums(T *a, T *blockSums, const uint numBlocks) {
    const uint blockId = blockId();
    if (blockId == 0 || blockId >= numBlocks) {
        return;
    }
    const uint i = blockSize * blockId + threadIdx.x;
    a[i] += blockSums[blockId];
}

template <typename T, uint blockSize>
void addBlockSums(T *a, T *blockSums, const uint numBlocks, const dim3 grid) {
    std::cout << "adding " << numBlocks << " blockSums to "
        << numBlocks << " blocks" << std::endl;
    addBlockSums<T, blockSize><<<grid, blockSize>>>(a, blockSums, numBlocks);
    check();
    sync();
}

template <typename T, uint blockSize>
void gpuScan(T *in, T *out, T *blockSums, const uint numBlocks) {
    const dim3 grid(numBlocks % MAX_GRID_SIZE, divUp(numBlocks, MAX_GRID_SIZE));
    scanBlocks<T, blockSize>(in, out, blockSums, numBlocks, grid);
    if (numBlocks == 1) {
        return;
    } else {
        // recurse
        const uint newNumBlocks = divUp(numBlocks, blockSize);
        T *newBlockSums = hipMalloc<T>(newNumBlocks);
        printf("scanning blockSums\n");
        gpuScan<T, blockSize>(blockSums, blockSums, newBlockSums, newNumBlocks);
        addBlockSums<T, blockSize>(out, blockSums, numBlocks, grid);
    }
}

template <typename T, uint blockSize>
double gpuScan(T *in, T *out, uint n) {
    clock_t start = clock();
    const uint numBlocks = divUp(n, blockSize);
    const uint paddedN = numBlocks * blockSize;
    T *dIn = cuMalloc<T>(paddedN);
    T *dOut = cuMalloc<T>(paddedN);
    T *blockSums = cuMalloc<T>(numBlocks);
    cuMemcpyTo<T>(dIn, in, n);
    gpuScan<T, blockSize>(dIn, dOut, blockSums, numBlocks);
    cuMemcpyFrom<T>(out, dOut, n);
    cuFree<T>(dIn);
    cuFree<T>(dOut);
    cuFree<T>(blockSums);
    return millis(start);
}

#define BLOCK_SIZE 512

#include "scanBase.cu"
